#include "hip/hip_runtime.h"
// clang-format off
#include "generalized_projection.h"
// clang-format on

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "common.cuh"
#include "utils.cuh"
#include "vec_utils.cuh"

namespace geomlib {
    namespace {
        template <int dim, typename scalar_t>
            __device__ inline void ComputeBarycentricGradient(
                    const scalar_t* e1, const scalar_t* e2, const scalar_t e1_dot_e2,
                    const scalar_t e1_norm_sqr, const scalar_t e2_norm_sqr, scalar_t* grad_w1,
                    scalar_t* grad_w2) {
                {
                    zero_out_vec<dim>(grad_w1);
                    add_vec<dim>(grad_w1, e1);
                    scalar_t tmp[dim];
                    scalar_times_vec<dim>(-e1_dot_e2 / e2_norm_sqr, e2, tmp);
                    add_vec<dim>(grad_w1, tmp);
                    scalar_t denom = e1_norm_sqr - e1_dot_e2 * e1_dot_e2 / e2_norm_sqr;
                    scalar_times_vec<dim>(1 / denom, grad_w1, grad_w1);
                }
                {
                    zero_out_vec<dim>(grad_w2);
                    add_vec<dim>(grad_w2, e2);
                    scalar_t tmp[dim];
                    scalar_times_vec<dim>(-e1_dot_e2 / e1_norm_sqr, e1, tmp);
                    add_vec<dim>(grad_w2, tmp);
                    scalar_t denom = e2_norm_sqr - e1_dot_e2 * e1_dot_e2 / e1_norm_sqr;
                    scalar_times_vec<dim>(1 / denom, grad_w2, grad_w2);
                }
            }

        template <int dim, typename scalar_t>
            __device__ void GeneralizedTriangleProjection(
                    const scalar_t* p, const scalar_t* v0, const scalar_t* e1,
                    const scalar_t* e2, const scalar_t e1_dot_e2, const scalar_t e1_norm_sqr,
                    const scalar_t e2_norm_sqr, const scalar_t* grad_w1,
                    const scalar_t* grad_w2,
                    // Below are results:
                    scalar_t* result_dist, scalar_t* result_w1, scalar_t* result_w2) {
                scalar_t w1, w2;
                {
                    scalar_t
                        p_minus_v0[dim];  // this array is unavoidable since we need to compute
                    // this for every pair of query point and face

                    minus_vec<dim>(p, v0, p_minus_v0);

                    w1 = dot_vec<dim>(grad_w1, p_minus_v0);
                    w2 = dot_vec<dim>(grad_w2, p_minus_v0);


                    if (0 <= 1 - w1 - w2 && 0 <= w1 && 0 <= w2) {
                        scalar_t p_proj[dim];
                        {
                            {
                                scalar_t w1_e1[dim];
                                scalar_times_vec<dim>(w1, e1, w1_e1);
                                scalar_times_vec<dim>(w2, e2, p_proj);
                                add_vec<dim>(p_proj, w1_e1);
                            }
                        }
                        *result_dist = distance_sqr_vec<dim>(p_minus_v0, p_proj);
                        *result_w1 = w1;
                        *result_w2 = w2;
                    } else {
                        // Project to three edges.
                        scalar_t w1_tmp[3];
                        scalar_t w2_tmp[3];

                        w1_tmp[0] = clamp01(dot_vec<dim>(p_minus_v0, e1) / e1_norm_sqr);
                        w2_tmp[0] = 0;

                        w2_tmp[1] = clamp01(dot_vec<dim>(p_minus_v0, e2) / e2_norm_sqr);
                        w1_tmp[1] = 0;

                        {
                            scalar_t numer = dot_vec<dim>(p_minus_v0, e2) -
                                dot_vec<dim>(p_minus_v0, e1) - e1_dot_e2 + e1_norm_sqr;
                            scalar_t denom = e1_norm_sqr + e2_norm_sqr - 2 * e1_dot_e2;
                            w2_tmp[2] = clamp01(numer / denom);
                            w1_tmp[2] = 1 - w2_tmp[2];
                        }

                        scalar_t best_dist = FLT_MAX;
                        int best_k = -1;

                        for (int k = 0; k < 3; k++) {
                            scalar_t p_proj[dim];
                            scalar_t w1_e1[dim];
                            scalar_times_vec<dim>(w1_tmp[k], e1, w1_e1);
                            scalar_times_vec<dim>(w2_tmp[k], e2, p_proj);
                            add_vec<dim>(p_proj, w1_e1);
                            scalar_t dist = distance_sqr_vec<dim>(p_minus_v0, p_proj);
                            if (dist < best_dist) {
                                best_dist = dist;
                                best_k = k;
                            }
                        }

                        *result_dist = best_dist;
                        *result_w1 = w1_tmp[best_k];
                        *result_w2 = w2_tmp[best_k];
                    }
                }
            }

        template <int dim, typename scalar_t>
            __device__ void GeneralizedTetrahedronProjection(
                    const scalar_t* p,              // D
                    const scalar_t* v0,             // D
                    const scalar_t* e_mat,          // 3xD,
                    const scalar_t* e_dot_mat,      // 3x3
                    const scalar_t* e_dot_inv_mat,  // 3x3
                    // Below are results:
                    scalar_t* result_dist,    // scalar
                    scalar_t* result_weights  // 3
                    ) {
                scalar_t p_minus_v0[dim];
                minus_vec<dim>(p, v0, p_minus_v0);

                scalar_t weights[3];
                {
                    scalar_t b[3];
                    for (int i = 0; i < 3; i++) {
                        b[i] = dot_vec<dim>(&e_mat[i * dim], p_minus_v0);
                    }
                    mat_vec_mult<3>(e_dot_inv_mat, b, weights);
                }

                // TODO: if adding break or has_negative_weight as a terminating condition,
                // then the result would be wrong. Is this a CUDA bug?
                bool has_negative_weight = false;
                {
                    scalar_t grad_wj[dim];
                    scalar_t grad_wk[dim];
                    scalar_t
                        tmp_weights[3];  // if using result_dist directly it would be buggy??
                    for (int i = 0; i < 3; i++) {
                        if (weights[i] < 0) {
                            int j = (i + 1) % 3;
                            int k = (i + 2) % 3;
                            ComputeBarycentricGradient<dim>(
                                    &e_mat[j * dim], &e_mat[k * dim], e_dot_mat[j * 3 + k],
                                    e_dot_mat[j * 3 + j], e_dot_mat[k * 3 + k], grad_wj, grad_wk);

                            GeneralizedTriangleProjection<dim>(
                                    p, v0, &e_mat[j * dim], &e_mat[k * dim], e_dot_mat[j * 3 + k],
                                    e_dot_mat[j * 3 + j], e_dot_mat[k * 3 + k], grad_wj, grad_wk,
                                    result_dist, &tmp_weights[j], &tmp_weights[k]);
                            tmp_weights[i] = 0;
                            has_negative_weight = true;
                            break;
                        }
                    }

                    if (has_negative_weight) {
                        copy_vec<3>(result_weights, tmp_weights);
                    }
                }

                if (!has_negative_weight) {
                    scalar_t weight_op = 1 - weights[0] - weights[1] - weights[2];
                    if (weight_op >= 0) {
                        for (int i = 0; i < 3; i++) {
                            result_weights[i] = weights[i];
                        }

                        // Compute distance.
                        scalar_t p_proj[dim];
                        zero_out_vec<dim>(p_proj);
                        for (int i = 0; i < 3; i++) {
                            scalar_t tmp[dim];
                            scalar_times_vec<dim>(result_weights[i], &e_mat[i * dim], tmp);
                            add_vec<dim>(p_proj, tmp);
                        }
                        *result_dist = distance_sqr_vec<dim>(p, p_proj);
                    } else {
                        // Project to the side opposite of v0.
                        scalar_t v3[dim];
                        plus_vec<dim>(v0, &e_mat[2 * dim], v3);
                        scalar_t e31[dim];
                        scalar_t e32[dim];
                        minus_vec<dim>(&e_mat[0 * dim], &e_mat[2 * dim], e31);
                        minus_vec<dim>(&e_mat[1 * dim], &e_mat[2 * dim], e32);
                        scalar_t e31_norm_sqr = dot_vec<dim>(e31, e31);
                        scalar_t e32_norm_sqr = dot_vec<dim>(e32, e32);
                        scalar_t e31_dot_e32 = dot_vec<dim>(e31, e32);

                        scalar_t grad_w31[dim];
                        scalar_t grad_w32[dim];
                        ComputeBarycentricGradient<dim>(e31, e32, e31_dot_e32, e31_norm_sqr,
                                e32_norm_sqr, grad_w31, grad_w32);

                        GeneralizedTriangleProjection<dim>(
                                p, v3, e31, e32, e31_dot_e32, e31_norm_sqr, e32_norm_sqr, grad_w31,
                                grad_w32, result_dist, &result_weights[0], &result_weights[1]);

                        result_weights[2] = 1 - result_weights[0] - result_weights[1];
                    }
                }
            }

        template <int dim, typename scalar_t>
            __global__ void GeneralizedTriangleProjectionKernel(
                    const scalar_t* __restrict__ points,  // P x D
                    const size_t num_faces,
                    // Pre-computed values:
                    const scalar_t* __restrict__ v0,           // FxD, vertex 0 of faces
                    const scalar_t* __restrict__ e1,           // FxD, vertex 1 - vertex 0
                    const scalar_t* __restrict__ e2,           // FxD
                    const scalar_t* __restrict__ e1_dot_e2,    // F
                    const scalar_t* __restrict__ e1_norm_sqr,  // F
                    const scalar_t* __restrict__ e2_norm_sqr,  // F
                    const scalar_t* __restrict__ grad_w1,      // FxD, dw1/dp on the face
                    const scalar_t* __restrict__ grad_w2,      // FxD
                    // Results:
                    scalar_t* __restrict__ result_dists, int* __restrict__ result_idxs,
                    scalar_t* __restrict__ result_w1, scalar_t* __restrict__ result_w2) {
                extern __shared__ char shared_buf[];
                scalar_t* min_dists = (scalar_t*)shared_buf;         // scalar_t[num_threads]
                size_t* min_tids = (size_t*)&min_dists[blockDim.x];  // size_t[num_threads]

                // Each block computing the projection of a single point.
                const size_t i = blockIdx.x;
                // Faces are divided evenly among the threads.
                const size_t tid = threadIdx.x;

                const scalar_t* p = &points[i * dim];  // point to project

                scalar_t min_dist = FLT_MAX;
                size_t min_idx = 0;
                scalar_t min_w1;
                scalar_t min_w2;

                for (int j = tid; j < num_faces; j += blockDim.x) {
                    scalar_t dist, w1, w2;
                    GeneralizedTriangleProjection<dim, scalar_t>(
                            p, &v0[j * dim], &e1[j * dim], &e2[j * dim], e1_dot_e2[j],
                            e1_norm_sqr[j], e2_norm_sqr[j], &grad_w1[j * dim], &grad_w2[j * dim],
                            &dist, &w1, &w2);

                    if (dist < min_dist) {
                        min_dist = dist;
                        min_idx = j;
                        min_w1 = w1;
                        min_w2 = w2;
                    }
                }
                min_dists[tid] = min_dist;
                min_tids[tid] = tid;
                __syncthreads();

                ReduceMin(min_dists, min_tids, tid);

                __syncthreads();

                // Finally thread with min_dist writes the result to the output.
                if (tid == min_tids[0]) {
                    result_dists[i] = min_dist;  // squared minimum distance
                    result_idxs[i] = min_idx;
                    assert(min_idx < num_faces);
                    result_w1[i] = min_w1;
                    result_w2[i] = min_w2;
                }
            }

        template <int dim, typename scalar_t>
            __global__ void GeneralizedTetrahedronProjectionKernel(
                    const scalar_t* __restrict__ points,  // P x D
                    const size_t num_tets,
                    // Pre-computed values:
                    const scalar_t* __restrict__ v0,             // TxD, vertex 0 of faces
                    const scalar_t* __restrict__ e_mat,          // Tx3xD, vertex i - vertex 0
                    const scalar_t* __restrict__ e_dot_mat,      // Tx3x3
                    const scalar_t* __restrict__ e_dot_inv_mat,  // Tx3x3
                    // Results:
                    scalar_t* __restrict__ result_dists, int* __restrict__ result_idxs,
                    scalar_t* __restrict__ result_weights  // Px3
                    ) {
                extern __shared__ char shared_buf[];
                scalar_t* min_dists = (scalar_t*)shared_buf;         // scalar_t[num_threads]
                size_t* min_tids = (size_t*)&min_dists[blockDim.x];  // size_t[num_threads]

                // Each block computing the projection of a single point.
                const size_t i = blockIdx.x;
                // Faces are divided evenly among the threads.
                const size_t tid = threadIdx.x;

                const scalar_t* p = &points[i * dim];  // point to project

                scalar_t min_dist = FLT_MAX;
                size_t min_idx = 0;
                scalar_t min_weights[3];

                for (int j = tid; j < num_tets; j += blockDim.x) {
                    scalar_t dist;
                    scalar_t weights[3];
                    GeneralizedTetrahedronProjection<dim, scalar_t>(
                            p, &v0[j * dim], &e_mat[j * 3 * dim], &e_dot_mat[j * 3 * 3],
                            &e_dot_inv_mat[j * 3 * 3], &dist, weights);

                    if (dist < min_dist) {
                        min_dist = dist;
                        min_idx = j;
                        copy_vec<3>(min_weights, weights);
                    }
                }
                min_dists[tid] = min_dist;
                min_tids[tid] = tid;
                __syncthreads();

                ReduceMin(min_dists, min_tids, tid);

                // Finally thread with min_dist writes the result to the output.
                if (tid == min_tids[0]) {
                    result_dists[i] = min_dist;  // squared minimum distance
                    result_idxs[i] = min_idx;
                    assert(min_idx < num_tets);
                    copy_vec<3>(&result_weights[i * 3], min_weights);
                }
            }
    }  // namespace

    template <int dim>
        std::vector<torch::Tensor> ComputeGeneralizedTriangleProjection(
                torch::Tensor points, const TriangularProjectionInfo& info) {
            CHECK_INPUT(points);

            TORCH_CHECK(points.size(1) == dim);
            TORCH_CHECK(info.dim == dim);
            TORCH_CHECK(points.dtype() == torch::kFloat32 ||
                    points.dtype() == torch::kFloat64);
            TORCH_CHECK(points.dtype() == info.v0.dtype());

            at::cuda::CUDAGuard device_guard{points.device()};
            hipStream_t stream = at::cuda::getCurrentCUDAStream();

            int num_points = points.size(0);
            int num_faces = info.num_faces;

            torch::Tensor result_dists =
                torch::zeros({num_points}, points.options()).contiguous();
            torch::Tensor result_idxs =
                torch::zeros({num_points}, points.options().dtype(torch::kInt32))
                .contiguous();
            torch::Tensor result_w1 =
                torch::zeros({num_points}, points.options()).contiguous();
            torch::Tensor result_w2 =
                torch::zeros({num_points}, points.options()).contiguous();

            size_t num_threads = 128;
            dim3 num_blocks(num_points);
            size_t shared_size =
                num_threads *
                ((points.dtype() == torch::kFloat32 ? sizeof(float) : sizeof(double)) +
                 sizeof(size_t));

            AT_DISPATCH_FLOATING_TYPES(
                    points.scalar_type(), "ComputeGeneralizedTriangleProjection", [&] {
                    GeneralizedTriangleProjectionKernel<dim, scalar_t>
                    <<<num_blocks, num_threads, shared_size, stream>>>(
                            points.contiguous().data_ptr<scalar_t>(),
                            static_cast<size_t>(num_faces),
                            info.v0.contiguous().data_ptr<scalar_t>(),
                            info.e1.contiguous().data_ptr<scalar_t>(),
                            info.e2.contiguous().data_ptr<scalar_t>(),
                            info.e1_dot_e2.contiguous().data_ptr<scalar_t>(),
                            info.e1_norm_sqr.contiguous().data_ptr<scalar_t>(),
                            info.e2_norm_sqr.contiguous().data_ptr<scalar_t>(),
                            info.grad_w1.contiguous().data_ptr<scalar_t>(),
                            info.grad_w2.contiguous().data_ptr<scalar_t>(),
                            result_dists.data_ptr<scalar_t>(), result_idxs.data_ptr<int>(),
                            result_w1.data_ptr<scalar_t>(), result_w2.data_ptr<scalar_t>());
                    });

            AT_CUDA_CHECK(hipGetLastError());

            return {result_dists, result_idxs.to(torch::kInt64), result_w1, result_w2};
        }

    template <int dim>
        std::vector<torch::Tensor> ComputeGeneralizedTetrahedronProjection(
                torch::Tensor points, torch::Tensor vertices, torch::Tensor tets) {
            CHECK_INPUT(points);
            CHECK_INPUT(vertices);
            CHECK_INPUT(tets);

            TORCH_CHECK(points.size(1) == dim);
            TORCH_CHECK(vertices.size(1) == dim);
            TORCH_CHECK(points.dtype() == torch::kFloat32 ||
                    points.dtype() == torch::kFloat64);
            TORCH_CHECK(tets.dtype() ==
                    torch::kInt64);  // torch requires 64-bit int for indexing
            TORCH_CHECK(points.dtype() == vertices.dtype());

            at::cuda::CUDAGuard device_guard{points.device()};
            hipStream_t stream = at::cuda::getCurrentCUDAStream();

            // Pre-compute reusable values.
            using namespace torch::indexing;
            std::vector<torch::Tensor> v_list;
            for (int i = 0; i < 4; i++) {
                v_list.push_back(vertices.index({tets.index({Slice(), i}), Slice()}));
            }
            std::vector<torch::Tensor> e_list;
            for (int i = 1; i < 4; i++) {
                e_list.push_back(v_list[i] - v_list[0]);
            }
            std::vector<torch::Tensor> e_dot_list;
            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 3; j++) {
                    e_dot_list.push_back((e_list[i] * e_list[j]).sum(-1));
                }
            }
            auto e_mat = torch::stack(e_list, 1);                              // Tx3xD
            auto e_dot_mat = torch::stack(e_dot_list, 1).reshape({-1, 3, 3});  // Tx3x3

            auto e_dot_inv_mat = torch::linalg_inv(e_dot_mat);  // Tx3x3

            size_t num_points = points.size(0);
            size_t num_tets = tets.size(0);

            torch::Tensor result_dists =
                torch::zeros({static_cast<int>(num_points)}, points.options())
                .contiguous();
            torch::Tensor result_idxs = torch::zeros({static_cast<int>(num_points)},
                    tets.options().dtype(torch::kInt32))
                .contiguous();
            torch::Tensor result_weights =
                torch::zeros({static_cast<int>(num_points), 3},
                        points.options())
                .contiguous();  // w1, w2, w3 of barycentric coordinates

            size_t num_threads = 128;
            dim3 num_blocks(num_points);
            size_t shared_size =
                num_threads *
                ((points.dtype() == torch::kFloat32 ? sizeof(float) : sizeof(double)) +
                 sizeof(size_t));

            AT_DISPATCH_FLOATING_TYPES(
                    points.scalar_type(), "ComputeGeneralizedTetrahedronProjection", [&] {
                    GeneralizedTetrahedronProjectionKernel<dim, scalar_t>
                    <<<num_blocks, num_threads, shared_size, stream>>>(
                            points.contiguous().data_ptr<scalar_t>(), num_tets,
                            v_list[0].contiguous().data_ptr<scalar_t>(),
                            e_mat.contiguous().data_ptr<scalar_t>(),
                            e_dot_mat.contiguous().data_ptr<scalar_t>(),
                            e_dot_inv_mat.contiguous().data_ptr<scalar_t>(),
                            result_dists.data_ptr<scalar_t>(), result_idxs.data_ptr<int>(),
                            result_weights.data_ptr<scalar_t>());
                    });

            AT_CUDA_CHECK(hipGetLastError());

            return {result_dists, result_idxs.to(torch::kInt64), result_weights};
        }

    // Explicit instantiations.
    template std::vector<torch::Tensor> ComputeGeneralizedTriangleProjection<3>(
            torch::Tensor points, const TriangularProjectionInfo& info);
    template std::vector<torch::Tensor> ComputeGeneralizedTriangleProjection<8>(
            torch::Tensor points, const TriangularProjectionInfo& info);
    template std::vector<torch::Tensor> ComputeGeneralizedTetrahedronProjection<3>(
            torch::Tensor points, torch::Tensor vertices, torch::Tensor tets);

}  // namespace geomlib
