#include "hip/hip_runtime.h"
#include "point_tri_mesh_test.h"

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "common.cuh"
#include "utils.cuh"

namespace geomlib {
    namespace {
        template <typename scalar_t>
            __global__ void PointTriMeshTestKernel(
                    const scalar_t* __restrict__ points,
                    const scalar_t* __restrict__ vertices,
                    const int* __restrict__ faces,
                    size_t num_faces,
                    const scalar_t* __restrict__ origin,
                    int* __restrict__ result_signs) {
                extern __shared__ char shared_buf[];
                int* windings = (int*)shared_buf;

                // Each block tests a single point.
                const size_t i = blockIdx.x;
                // Faces are divided evenly among the threads.
                const size_t tid = threadIdx.x;
                auto origin_p = MakeVec3<scalar_t>::FromRaw(origin);
                int winding = 0;
                for (size_t j = tid; j < num_faces; j += blockDim.x) {
                    auto p = &points[i * 3];
                    auto v0 = &vertices[3 * faces[3 * j]];
                    auto v1 = &vertices[3 * faces[3 * j + 1]];
                    auto v2 = &vertices[3 * faces[3 * j + 2]];
                    if (IsPointInTetrahedron<scalar_t>(
                                MakeVec3<scalar_t>::FromRaw(p), MakeVec3<scalar_t>::FromRaw(v0),
                                MakeVec3<scalar_t>::FromRaw(v1), MakeVec3<scalar_t>::FromRaw(v2),
                                origin_p)) {
                        winding += 1;
                    }
                }
                windings[tid] = winding;
                __syncthreads();

                // Reduction.
                for (size_t s = blockDim.x / 2; s > 32; s >>= 1) {
                    if (tid < s) {
                        windings[tid] += windings[tid + s];
                    }
                    __syncthreads();
                }

                if (tid < 32) {
                    WarpReduceSum<int>(windings, tid);
                }

                if (tid == 0) {
                    int sgn = windings[0] % 2 == 0 ? 1 : -1;
                    result_signs[i] = sgn;
                }
            }
    }  // namespace

    torch::Tensor PointTriMeshTest(torch::Tensor points, torch::Tensor vertices,
            torch::Tensor faces, torch::Tensor origin) {
        CHECK_INPUT(points);
        CHECK_INPUT(vertices);
        CHECK_INPUT(faces);
        CHECK_INPUT(origin);

        TORCH_CHECK(points.size(1) == 3);
        TORCH_CHECK(vertices.size(1) == 3);
        TORCH_CHECK(faces.size(1) == 3);
        TORCH_CHECK(origin.size(0) == 3);
        TORCH_CHECK(points.dtype() == torch::kFloat32 ||
                points.dtype() == torch::kFloat64);
        TORCH_CHECK(origin.dtype() == points.dtype());
        TORCH_CHECK(faces.dtype() == torch::kInt32 || faces.dtype() == torch::kInt64);
        TORCH_CHECK(points.dtype() == vertices.dtype());

        at::cuda::CUDAGuard device_guard{points.device()};
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        if (faces.dtype() == torch::kInt64) {
            faces = faces.to(torch::kInt32);
        }

        size_t num_points = points.size(0);
        size_t num_faces = faces.size(0);
        torch::Tensor result_signs =
            torch::zeros({static_cast<int>(num_points)}, faces.options()).contiguous();

        size_t num_threads = 128;
        dim3 num_blocks(num_points);
        size_t shared_size = num_threads * sizeof(int);
        AT_DISPATCH_FLOATING_TYPES(points.scalar_type(), "PointTriMeshTest", [&] {
                PointTriMeshTestKernel<scalar_t>
                <<<num_blocks, num_threads, shared_size, stream>>>(
                        points.contiguous().data_ptr<scalar_t>(),
                        vertices.contiguous().data_ptr<scalar_t>(),
                        faces.contiguous().data_ptr<int>(), num_faces,
                        origin.contiguous().data_ptr<scalar_t>(),
                        result_signs.data_ptr<int>());
                });

        AT_CUDA_CHECK(hipGetLastError());

        return result_signs;
    }
}  // namespace geomlib
