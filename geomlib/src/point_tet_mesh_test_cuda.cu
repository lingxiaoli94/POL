#include "hip/hip_runtime.h"
#include "point_tet_mesh_test.h"

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "common.cuh"
#include "utils.cuh"

namespace geomlib {
    namespace {
        template <typename scalar_t>
            __global__ void PointTetMeshTestKernel(const scalar_t* __restrict__ points,
                    const scalar_t* __restrict__ vertices,
                    const int* __restrict__ tets,
                    size_t num_tets,
                    int* __restrict__ result_signs) {
                extern __shared__ char shared_buf[];
                int* windings = (int*)shared_buf;

                // Each block tests a single point.
                const size_t i = blockIdx.x;
                // Tets are divided evenly among the threads.
                const size_t tid = threadIdx.x;

                int winding = 0;
                for (size_t j = tid; j < num_tets; j += blockDim.x) {
                    auto p = &points[i * 3];
                    auto v0 = &vertices[3 * tets[4 * j]];
                    auto v1 = &vertices[3 * tets[4 * j + 1]];
                    auto v2 = &vertices[3 * tets[4 * j + 2]];
                    auto v3 = &vertices[3 * tets[4 * j + 3]];
                    if (IsPointInTetrahedron<scalar_t>(
                                MakeVec3<scalar_t>::FromRaw(p), MakeVec3<scalar_t>::FromRaw(v0),
                                MakeVec3<scalar_t>::FromRaw(v1), MakeVec3<scalar_t>::FromRaw(v2),
                                MakeVec3<scalar_t>::FromRaw(v3))) {
                        winding += 1;
                    }
                }
                windings[tid] = winding;
                __syncthreads();

                // Reduction.
                for (size_t s = blockDim.x / 2; s > 32; s >>= 1) {
                    if (tid < s) {
                        windings[tid] += windings[tid + s];
                    }
                    __syncthreads();
                }

                if (tid < 32) {
                    WarpReduceSum<int>(windings, tid);
                }

                if (tid == 0) {
                    int sgn = windings[0] > 0 ? -1 : 1;
                    result_signs[i] = sgn;
                }
            }
    }  // namespace

    torch::Tensor PointTetMeshTest(torch::Tensor points, torch::Tensor vertices,
            torch::Tensor tets) {
        CHECK_INPUT(points);
        CHECK_INPUT(vertices);
        CHECK_INPUT(tets);

        TORCH_CHECK(points.size(1) == 3);
        TORCH_CHECK(vertices.size(1) == 3);
        TORCH_CHECK(tets.size(1) == 4);
        TORCH_CHECK(points.dtype() == torch::kFloat32 ||
                points.dtype() == torch::kFloat64);
        TORCH_CHECK(tets.dtype() == torch::kInt32 || tets.dtype() == torch::kInt64);
        TORCH_CHECK(points.dtype() == vertices.dtype());

        at::cuda::CUDAGuard device_guard{points.device()};
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        if (tets.dtype() == torch::kInt64) {
            tets = tets.to(torch::kInt32);
        }

        size_t num_points = points.size(0);
        size_t num_tets = tets.size(0);
        torch::Tensor result_signs =
            torch::zeros({static_cast<int>(num_points)}, tets.options()).contiguous();

        size_t num_threads = 128;
        dim3 num_blocks(num_points);
        size_t shared_size = num_threads * sizeof(int);
        AT_DISPATCH_FLOATING_TYPES(points.scalar_type(), "PointTetMeshTest", [&] {
                PointTetMeshTestKernel<scalar_t>
                <<<num_blocks, num_threads, shared_size, stream>>>(
                        points.contiguous().data_ptr<scalar_t>(),
                        vertices.contiguous().data_ptr<scalar_t>(),
                        tets.contiguous().data_ptr<int>(), num_tets,
                        result_signs.data_ptr<int>());
                });

        AT_CUDA_CHECK(hipGetLastError());

        return result_signs;
    }
}  // namespace geomlib
